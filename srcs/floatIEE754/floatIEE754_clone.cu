#include "float754.h"

t_float754 *float754_clone(t_float754 *a) {
	if (a == NULL) {
		return (NULL);
	}

	int size = sizeof(t_float754) + a->sizebyte;
	t_float754 *b = (t_float754*)malloc(size);
	if (b == NULL) {
		return (NULL);
	}
	memcpy(b, a, size);
	return (b);
}