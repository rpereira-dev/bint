# include "float754.h"

t_float754 *float754_negate(t_float754 *f) {

	if (f != NULL) {
		((char*)(f + 1))[0] ^= (1 << 7);
	}

	return (f);
}
