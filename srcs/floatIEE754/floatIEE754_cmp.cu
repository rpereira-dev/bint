# include "float754.h"

int float754_cmp(t_float754 *a, t_float754 *b) {

	//if we are comparing the same pointer address
	if (a == b) {
		return (0);
	}
	//else if a != b

	//if a is NULL, b isnt NULL (because a != b)
	if (a == NULL) {
		//if b is negative, return -1, +1 if positive
		return (float754_sign(b));
	}

	//if b is NULL, a isnt NULL (because a != b)
	if (b == NULL) {
		//if a is negative, return -1, +1 if positive
		return (float754_sign(a));
	}

	
	return (0);

	//TODO
}