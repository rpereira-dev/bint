# include "float754.h"

t_float754 *float754_add(t_float754 *dst, t_float754 *a, t_float754 *b) {

	//assume that a and b are positive

	//always have a > b
	if (float754_cmp(a, b) < 0) {
		t_float754 *tmp = a;
		a = b;
		b = tmp;
	}

	//so here we always have b < a
	

	(void)dst;
	(void)a;
	(void)b;
	return (dst);
}
