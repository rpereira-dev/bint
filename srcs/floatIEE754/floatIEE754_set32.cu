# include "float754.h"

//this method will not work on float with an exposant > 1 byte, FIX ME!
t_float754 *float754_set32(t_float754 *F, float f) {

	//reset the float
	memset(F + 1, 0, F->sizebyte);

	//check endianess so we always work in little endian
	f = ensure_float_endianess(f);

begin_exposant:

	//get the raw bits without the sign
	char *bytes = (char*)&f;
	int raw = *((int*)bytes) << 1;

	//get the real exposant value
	char exposant = *((char*)&raw) + 127;

	//copy the real exposant value
	char *dst = (char*)(F + 1);
	dst[F->exposantbyte - 1] = exposant;

	//apply offset on the copied exposant value
	//notice that this addition is done by doing it with integer to reduce instructions

	//small offset byte
	char soffbyte = (1 << 7) - 1; //01111111
	//big offset byte
	char boffbyte = -1; //11111111

	//small offset integer, this trick is done to handle different endianness
	int soffint; //01111111 11111111 11111111 11111111
	char *soffintaddr = (char*)&soffint;
	soffintaddr[0] = soffbyte;
	soffintaddr[1] = boffbyte;
	soffintaddr[2] = boffbyte;
	soffintaddr[3] = boffbyte;

	//big offset integer
	int boffint = -1; //11111111 11111111 11111111 11111111

	//iterator
	int i;
	//add integers by integers
	if (F->exposantbyte >= 4) {
		i = F->exposantbyte - 4;
		while (true) {

			//the exposant size is a multiple of 4, nice, we are done
			if (i == 0) {
				int *addr = (int*)(dst + i);
				*addr = *addr - soffint;
				goto end_exposant;
			}

			//we are not done yet, add the 4 byte part
			int *addr = (int*)(dst + i);
			*addr = *addr - boffint;

			//are we done? if so, stop adding them 4 by 4
			if (i < 4) {
				break ;
			}
		}
	} else {
		i = F->exposantbyte - 1;
	}

	//add the remaining byte one by one
	while (i > 0) {

		dst[i] = dst[i] - boffbyte;
		--i;
	}

	//add the final byte
	dst[0] = dst[0] - soffbyte;

end_exposant:

begin_mantissa:
	dst = dst + F->exposantbyte;
	char *mantissa = ((char*)&raw) + 1; //offset of the exposant
	memcpy(dst + F->exposantbyte, mantissa, 3);

	return (F);
}