# include "float754.h"

void float754_dump(t_float754 *f) {

	char *addr = (char*)(f + 1);
	printf("exposant:{");
	bindump(addr, f->exposantbyte);
	printf("} , mantissa:{");
	bindump(addr + f->exposantbyte, f->mantissabyte);
	printf("}\n");
}