# include "float754.h"

void float754_dump(t_float754 *f) {

	if (f == NULL) {
		printf("(null)\n");
	} else {

		char *addr = (char*)(f + 1);
		char *end = addr + f->sizebyte;
		char buffer[9];
		buffer[8] = 0;

		while (addr < end) {
			int i;
			for (i = 0 ; i < 8 ; i++) {
				buffer[i] = ((*addr) & (1 << (7 - i))) ? '1' : '0';
			}
			printf("%s ", buffer);
			++addr;
		}
		printf("\n");
	}
}