# include "float754.h"

static int str_is_zero(unsigned char const *str) {

	//if the string is null
	if (str == NULL) {
		return (NULL);
	}

	//if the string only contains 0 char
	//TODO optimize this on this model: http://www.stdlib.net/~colmmacc/strlen.c.html
	const unsigned char *s;
	for (s = str; *s == '0'; ++s);
	return (*s == 0);
}

t_float754 *float754_set(t_float754 *f, unsigned char const *str) {

	//if the given float is NULL
	if (f == NULL) {
		//return NULL because we cannot know how to interpert the string
		return (NULL);
	}

	//if we want to set 0
	if (str_is_zero(str)) {
		//set it to zero
		memset(f + 1, 0, f->sizebyte);
		return (f);
	}

	//here we have a float with the right sizes
	const unsigned char *s = str;
	unsigned char *addr = (unsigned char*)(f + 1);
	int i = 0;

	memset(addr, 0, f->sizebyte);

	//for each byte
	while (i < f->sizebyte && *str) {
		addr[i] |= (*s++ == '1') ? (1 << 7) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 6) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 5) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 4) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 3) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 2) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 1) : 0;
		addr[i] |= (*s++ == '1') ? (1 << 0) : 0;

		if (*s == ' ') {
			++s;
		}

		//jump to next byte
		++i;
	}
	return (f);
}

static int endianess() {
	union {
		int i;
		char c[4];
	} bint = {0x01020304};
	return (bint.c[0] == 1); 
}

static void swap_char(char *a, char *b) {
	char c = *a;
	*a = *b;
	*b = c;
}

t_float754 *float754_set32(t_float754 *F, float f) {

	char *bytes = (char*)&f;
	if (endianess() == 0) {
		swap_char(bytes, bytes + 3);	
		swap_char(bytes + 1, bytes + 2);	
	}

	memcpy(F + 1, &f, sizeof(float));
	return (F);
}