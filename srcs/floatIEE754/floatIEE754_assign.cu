# include "float754.h"

t_float754 *float754_assign(t_float754 *dst, t_float754 *a) {

	//if dst is NULL, clone a
	if (dst == NULL) {
		return (float754_clone(a));
	}

	//if we want to assign 0
	if (a == NULL) {
		memset(dst + 1, 0, dst->sizebyte);
		return (dst);
	}

	//if sizes are different, return null
	if (memcmp(dst, a, sizeof(t_float754)) != 0) {
		return (NULL);
	}

	memcpy(dst + 1, a + 1, a->sizebyte);
	return (dst);
}