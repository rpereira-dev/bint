# include "float754.h"

t_float754 *float754_assign(t_float754 *dst, t_float754 *a) {

	//if dst is NULL, clone a
	if (dst == NULL) {
		return (float754_clone(a));
	}

	//if we want to assign 0
	if (a == NULL) {
		dst->sign = 0;
		memset(dst + 1, 0, dst->sizebyte);
		return (dst);
	}

	//wrong sizes
	if (dst->exposantbyte != a->exposantbyte || dst->mantissabyte != a->exposantbyte) {
		return (NULL);
	}

	memcpy(dst + 1, a + 1, a->sizebyte);
	return (dst);
}