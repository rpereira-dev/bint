# include "float754.h"

int float754_sign(t_float754 *f) {
	
	if (f == NULL) {
		return (0);
	}

	int sign = *((char*)(f + 1)) & (1 << 7);
	return (sign == 0 ? 1 : -1);
}
