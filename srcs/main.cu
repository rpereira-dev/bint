#include "hip/hip_runtime.h"
#include "fractal.h"
#include <time.h>

void testBint() {

	int avalue = 42;
	int bvalue = 10;

	t_bint *a = bint_new(10000000);
	t_bint *b = bint_new(10000000);
	t_bint *r = bint_new(10000000);

	bint_set32(a, avalue);
	bint_set32(b, bvalue);

	printf("%10d: ", avalue), bint_dump(a), printf("\n");
	printf("%10d: ", bvalue), bint_dump(b), printf("\n");

	clock_t t = clock();
	r = bint_add_dst(&r, a, b);
	printf("%u\n", clock() - t);
	printf("%10d: ", avalue + bvalue), bint_dump(r), printf("\n");

	free(a);
	free(b);
	free(r);
}

int main(void) {
	
	//testFloatIEE754();
	testBint();

	return (EXIT_SUCCESS);
}
