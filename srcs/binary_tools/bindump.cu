#include "binary_tools.h"

void bindump(void *data, int len) {

	if (data == NULL) {
		printf("(null)\n");
	} else if (len > 0) {

		char *addr = (char*)data;
		char *end = addr + len;
		char buffer[9];
		buffer[8] = 0;

		while (true) {

			buffer[0] = ((*addr) & (1 << 7)) ? '1' : '0';
			buffer[1] = ((*addr) & (1 << 6)) ? '1' : '0';
			buffer[2] = ((*addr) & (1 << 5)) ? '1' : '0';
			buffer[3] = ((*addr) & (1 << 4)) ? '1' : '0';
			buffer[4] = ((*addr) & (1 << 3)) ? '1' : '0';
			buffer[5] = ((*addr) & (1 << 2)) ? '1' : '0';
			buffer[6] = ((*addr) & (1 << 1)) ? '1' : '0';
			buffer[7] = ((*addr) & (1 << 0)) ? '1' : '0';

			printf(buffer);
			++addr;
			if (addr < end) {
				printf(" ");
			} else {
				break ;
			}
		}
	}
}
