
#include <hip/hip_runtime.h>

int endianness() {
	union {
		int i;
		char c[4];
	} bint = {0x01020304};
	return (bint.c[0] == 1); 
}

static void swap_char(char *a, char *b) {
	char c = *a;
	*a = *b;
	*b = c;
}

float btools_swap_float_endian(float f) {
	char *bytes = (char*)&f;
	swap_char(bytes, bytes + 3);	
	swap_char(bytes + 1, bytes + 2);	
	return (f);
}

int btools_swap_int_endian(int i) {
	char *bytes = (char*)&i;
	swap_char(bytes, bytes + 3);	
	swap_char(bytes + 1, bytes + 2);
	return (i);
}

unsigned int btools_swap_unsigned_int_endian(unsigned int i) {
	char *bytes = (char*)&i;
	swap_char(bytes, bytes + 3);	
	swap_char(bytes + 1, bytes + 2);
	return (i);
}
