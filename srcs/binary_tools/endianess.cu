
#include <hip/hip_runtime.h>

int endianess() {
	union {
		int i;
		char c[4];
	} bint = {0x01020304};
	return (bint.c[0] == 1); 
}

static void swap_char(char *a, char *b) {
	char c = *a;
	*a = *b;
	*b = c;
}

float ensure_float_endianess(float f) {
	char *bytes = (char*)&f;
	if (endianess() == 0) {
		swap_char(bytes, bytes + 3);	
		swap_char(bytes + 1, bytes + 2);	
	}
	return (f);
}

int ensure_int_endianess(int i) {
	char *bytes = (char*)&i;
	if (endianess() == 0) {
		swap_char(bytes, bytes + 3);	
		swap_char(bytes + 1, bytes + 2);	
	}
	return (i);
}
