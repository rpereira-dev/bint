#include "hip/hip_runtime.h"
#include "bint.h"

/** a slow addition version which add bits by bits, but handle overflowing */

/*
#define PROCESS_BIT(i) total = BITSET(*abits, i) + BITSET(*bbits, i) + reminder; \
			if (total == 0) {\
				UNSETBIT(*dstbits, i);\
			} else if (total == 1 && reminder == 0) {\
				SETBIT(*dstbits, i);\
			} else if (total == 1 && reminder != 0) {\
				reminder = 0;\
				SETBIT(*dstbits, i);\
			} else if (total == 2 && reminder == 0) {\
				reminder = 1;\
				UNSETBIT(*dstbits, i);\
			} else if (total == 2 && reminder != 0) {\
				reminder = 0;\
				UNSETBIT(*dstbits, i);\
			} else if (total == 3) {\
				reminder = 1,\
				SETBIT(*dstbits, i);\
			}

static void bint_add_bits_by_bits_dst(t_bint *dst, t_bint *a, t_bint *b) {

	dst->sign = a->sign;
	int *abits = (int*)(a->bits + a->size);
	int *bbits = (int*)(b->bits + b->size);
	int *dstbits = (int*)(dst->bits + dst->size);
	int reminder = 0;
	int total = 0;
	int *end = (int*)a->bits;

	while (--abits >= end) {

		--bbits;
		--dstbits;

		PROCESS_BIT(0);
		PROCESS_BIT(1);
		PROCESS_BIT(2);
		PROCESS_BIT(3);
		PROCESS_BIT(4);
		PROCESS_BIT(5);
		PROCESS_BIT(6);
		PROCESS_BIT(7);

		PROCESS_BIT(8);
		PROCESS_BIT(9);
		PROCESS_BIT(10);
		PROCESS_BIT(11);
		PROCESS_BIT(12);
		PROCESS_BIT(13);
		PROCESS_BIT(14);
		PROCESS_BIT(15);

		PROCESS_BIT(16);
		PROCESS_BIT(17);
		PROCESS_BIT(18);
		PROCESS_BIT(19);
		PROCESS_BIT(20);
		PROCESS_BIT(21);
		PROCESS_BIT(22);
		PROCESS_BIT(23);

		PROCESS_BIT(24);
		PROCESS_BIT(25);
		PROCESS_BIT(26);
		PROCESS_BIT(27);
		PROCESS_BIT(28);
		PROCESS_BIT(29);
		PROCESS_BIT(30);
		PROCESS_BIT(31);
	}
}
*/

/** add the two numbers, assuming they have the same sign and a >= b */
static void _bint_add_dst_raw(t_bint *dst, t_bint *a, t_bint *b) {

	unsigned int *awords = a->words + a->size;
	unsigned int *bwords = b->words + b->size;
	unsigned int *dstwords = dst->words + dst->size;
	unsigned int reminder = 0;

	//add the two integers
	do {
		*(--dstwords) = *(--awords) + *(--bwords) + reminder;
		reminder = *dstwords < *awords || *dstwords < *bwords;
	} while (bwords >= b->last_word_set);

	//if they are remaining bits to add in 'a' and a reminder
	if (awords >= b->last_word_set && reminder) {
		*(--dstwords) = *(--awords) + reminder;
	}

	//if they are STILL remaining bits to add in 'a'
	while (awords >= b->last_word_set) {
		*(--dstwords) = *(--awords);
	}

	dst->last_word_set = dstwords + 1;
}

/** add two integer a, b, with the same sizes, they are assumed non-both NULL */
static void _bint_add_dst(t_bint *dst, t_bint *a, t_bint *b) {

	//if one is NULL, return a copy of the other
	if (a == NULL || a->sign == 0) {
		bint_copy(dst, b);
		return ;
	}
	if (b == NULL || b->sign == 0) {
		bint_copy(dst, a);
		return ;
	}

	//if a < b
	if (bint_cmp(a, b) < 0) {
		//swap them
		t_bint *tmp = a;
		a = b;
		b = tmp;
	}

	//if both negative or both positive
	if (a->sign == -1 && b->sign == 1) {
		//TODO SUB b - abs(a)
		return ;
	} else if (a->sign == 1 && b->sign == -1) {
		//TODO sub fast a - abs(b)
		return ;
	} else {
		dst->sign = a->sign;
		_bint_add_dst_raw(dst, a, b);
	}
}

t_bint *bint_add(t_bint *a, t_bint *b) {
	return (bint_add_dst(NULL, a, b));
}

t_bint *bint_add_dst(t_bint **dst, t_bint *a, t_bint *b) {

	//if a and b are 0
	if ((a == NULL || a->sign == 0) && (b == NULL || b->sign == 0)) {
		//return 0
		return (NULL);
	}

	//the size to store the result
	int size = a == NULL ? b->size : b == NULL ? a->size : a->size > b->size ? a->size : b->size;

	//the return value
	t_bint *r;

	//if dst is NULL
	if (dst == NULL) {
		//create a new integer
		r = bint_new(size);
	} else if ((*dst) == NULL || (*dst)->size < size) {
		//free it
		free(*dst);
		//allocate the return value
		r = bint_new(size);
		//set the return value to dst
		*dst = r;
	} else {
		r = *dst;
	}

	_bint_add_dst(r, a, b);

	return (r);
}

