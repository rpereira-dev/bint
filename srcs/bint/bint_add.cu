#include "hip/hip_runtime.h"
#include "bint.h"

/** a slow addition version which add bits by bits, but handle overflowing */

/*
#define PROCESS_BIT(i) total = BITSET(*abits, i) + BITSET(*bbits, i) + reminder; \
			if (total == 0) {\
				UNSETBIT(*dstbits, i);\
			} else if (total == 1 && reminder == 0) {\
				SETBIT(*dstbits, i);\
			} else if (total == 1 && reminder != 0) {\
				reminder = 0;\
				SETBIT(*dstbits, i);\
			} else if (total == 2 && reminder == 0) {\
				reminder = 1;\
				UNSETBIT(*dstbits, i);\
			} else if (total == 2 && reminder != 0) {\
				reminder = 0;\
				UNSETBIT(*dstbits, i);\
			} else if (total == 3) {\
				reminder = 1,\
				SETBIT(*dstbits, i);\
			}

static void bint_add_bits_by_bits_dst(t_bint *dst, t_bint *a, t_bint *b) {

	dst->sign = a->sign;
	int *abits = (int*)(a->bits + a->size);
	int *bbits = (int*)(b->bits + b->size);
	int *dstbits = (int*)(dst->bits + dst->size);
	int reminder = 0;
	int total = 0;
	int *end = (int*)a->bits;

	while (--abits >= end) {

		--bbits;
		--dstbits;

		PROCESS_BIT(0);
		PROCESS_BIT(1);
		PROCESS_BIT(2);
		PROCESS_BIT(3);
		PROCESS_BIT(4);
		PROCESS_BIT(5);
		PROCESS_BIT(6);
		PROCESS_BIT(7);

		PROCESS_BIT(8);
		PROCESS_BIT(9);
		PROCESS_BIT(10);
		PROCESS_BIT(11);
		PROCESS_BIT(12);
		PROCESS_BIT(13);
		PROCESS_BIT(14);
		PROCESS_BIT(15);

		PROCESS_BIT(16);
		PROCESS_BIT(17);
		PROCESS_BIT(18);
		PROCESS_BIT(19);
		PROCESS_BIT(20);
		PROCESS_BIT(21);
		PROCESS_BIT(22);
		PROCESS_BIT(23);

		PROCESS_BIT(24);
		PROCESS_BIT(25);
		PROCESS_BIT(26);
		PROCESS_BIT(27);
		PROCESS_BIT(28);
		PROCESS_BIT(29);
		PROCESS_BIT(30);
		PROCESS_BIT(31);
	}
}
*/

/** add the two numbers, assuming they have the same sign and a >= b */
static void _bint_add_dst_raw(t_bint *dst, t_bint *a, t_bint *b) {

	unsigned int *awords = a->words + a->size;
	unsigned int *bwords = b->words + b->size;
	unsigned int *dstwords = dst->words + dst->size;
	unsigned int reminder = 0;

	//add the two integers
	do {
		*(--dstwords) = *(--awords) + *(--bwords) + reminder;
		reminder = *dstwords < *awords || *dstwords < *bwords;
	} while (bwords >= b->last_word_set);

	//if they are remaining bits to add in 'a' and a reminder
	if (awords >= b->last_word_set && reminder) {
		*(--dstwords) = *(--awords) + reminder;
	}

	//if they are STILL remaining bits to add in 'a'
	while (awords >= b->last_word_set) {
		*(--dstwords) = *(--awords);
	}

	dst->last_word_set = dstwords + 1;
}

t_bint *bint_add(t_bint *a, t_bint *b) {
	return (bint_add_dst(NULL, a, b));
}

t_bint *bint_add_dst(t_bint **dst, t_bint *a, t_bint *b) {

	//if a and b are 0
	if (bint_is_zero(a) && bint_is_zero(b)) {
		//return 0
		return (BINT_ZERO);
	}

	//the size to store the result
	int size = a == NULL ? b->size : b == NULL ? a->size : a->size > b->size ? a->size : b->size;

	//ensure that 'dst' bint has the given size
	t_bint * r = bint_ensure_size(dst, size);

	//if allocation failed
	if (r == NULL) {
		return (NULL);
	}

	//do the addition, r has now a correct size to store the result

	//if one is NULL, return a copy of the other
	if (a == NULL || a->sign == 0) {
		bint_copy(r, b);
	} else if (b == NULL || b->sign == 0) {
		bint_copy(r, a);
	} else {

		//compare the two integers
		int cmp = bint_cmp(a, b);
		
		//if a == b, then return 2 * a
		if (cmp == 0) {
			//dst = a << 1 = 2 * a
			bint_shift_left_dst(dst, a, 1);
		} else {

			//else, if a < b
			if (cmp < 0) {
				//swap them, so we always then have a > b
				t_bint *tmp = a;
				a = b;
				b = tmp;
			}
			
			//' a + (-b) ' becomes ' a - b '
			//notice that the case '(-a) + b ' becoming ' b - a' is impossible here: a > b
			if (a->sign == 1 && b->sign == -1) {
				//TODO : a - abs(b)
			} else {
				//a and b have the same size, and a > b
				//set the sign
				r->sign = a->sign;

				//finally do the addition
				_bint_add_dst_raw(r, a, b);
			}

		}
	}

	//return the result
	return (r);
}

