#include "bint.h"

int bint_cmp(t_bint *a, t_bint *b) {
	//if a and b are NULL
	if (a == NULL && b == NULL) {
		//0 == 0
		return (1);
	}

	//if a is NULL, then b isnt NULL
	if (a == NULL) {
		return (b->sign);
	}

	//if b is NULL, then a isnt NULL
	if (b == NULL) {
		return (a->sign);
	}

	//if they are both non NULL
	//compare sign
	if (a->sign > b->sign) {
		return (1);
	}
	if (a->sign < b->sign) {
		return (-1);
	}

	//now we ensure that a's sign equals b's one
	int sign = a->sign;

	//compare word set (as they now have the same sign)
	unsigned int aset = a->last_word_set - a->words;
	unsigned int bset = b->last_word_set - b->words;

	//if a is shorter than b
	if (aset < bset) {
		//return -sign(a) , e.g, a = 4, b = 47, return -sign(4) = -1
		return (-sign);
	}

	if (aset > bset) {
		return (sign);
	}

	//else they have the same size
	//get the biggest word set
	unsigned int *aword = a->last_word_set;
	unsigned int *bword = b->last_word_set;

	//the end of the loop, as we know a and b have the same size also
	unsigned int *aend = a->words + a->size;

	do {
		if (*aword < *bword) {
			return (-sign);
		} else if (*aword > *bword) {
			return (sign);
		}
		++aword;
		++bword;
	} while (aword < aend);

	puts("OK MEC");

	//else a strictly equals b
	return (0);
}