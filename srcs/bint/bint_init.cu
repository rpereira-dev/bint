#include "bint.h"

t_bint * bint_zero;
t_bint * bint_two;
t_bint *bint_eight;
t_bint * bint_ten;

void(*bint_set64)(t_bint *, long int );
void(*bint_set32)(t_bint *, int );
void(*bint_set16)(t_bint *, short );

/** operations, allocate a new integer */
t_bint *(*bint_add)(t_bint *a, t_bint *b);
t_bint *(*bint_sub)(t_bint *a, t_bint *b);
t_bint *(*bint_mult)(t_bint *a, t_bint *b);
t_bint *(*bint_div)(t_bint *a, t_bint *b);
t_bint *(*bint_mod)(t_bint *a, t_bint *b);
t_bint **(*bint_divmod)(t_bint *a, t_bint *b);

/** operation with a destination pointer address given as argument */
/** arguments are : (dst, a, b) : dst = a OP b */
void (*bint_add_dst)(t_bint *dst, t_bint *a, t_bint *b);
void (*bint_sub_dst)(t_bint *dst, t_bint *a, t_bint *b);
void (*bint_mult_dst)(t_bint *dst, t_bint *a, t_bint *b);
void (*bint_div_dst)(t_bint *dst, t_bint *a, t_bint *b);
void (*bint_mod_dst)(t_bint *dst, t_bint *a, t_bint *b);
void (*bint_divmod_dst)(t_bint **dst, t_bint *a, t_bint *b);


int bint_init() {

	int endian = endianness();

	if (endian == BIG_ENDIAN) {
		//big endian function set
		bint_set64 = bint_set64_be;
		bint_set32 = bint_set32_be;
		bint_set16 = bint_set16_be;

		bint_add = bint_add_be;
		bint_sub = bint_sub_be;
		bint_mult = bint_mult_be;
		bint_div = bint_div_be;
		bint_mod = bint_mod_be;
		bint_divmod = bint_divmod_be;

		bint_add_dst = bint_add_dst_be;
		bint_sub_dst = bint_sub_dst_be;
		bint_mult_dst = bint_mult_dst_be;
		bint_div_dst = bint_div_dst_be;
		bint_mod_dst = bint_mod_dst_be;
		bint_divmod_dst = bint_divmod_dst_be;
	} else {
		//little endian function set
		bint_set64 = bint_set64_le;
		bint_set32 = bint_set32_le;
		bint_set16 = bint_set16_le;

		bint_add = bint_add_le;
		bint_sub = bint_sub_le;
		bint_mult = bint_mult_le;
		bint_div = bint_div_le;
		bint_mod = bint_mod_le;
		bint_divmod = bint_divmod_le;

		bint_add_dst = bint_add_dst_le;
		bint_sub_dst = bint_sub_dst_le;
		bint_mult_dst = bint_mult_dst_le;
		bint_div_dst = bint_div_dst_le;
		bint_mod_dst = bint_mod_dst_le;
		bint_divmod_dst = bint_divmod_dst_le;
	}


	//constants
	bint_zero = bint_new32(0);
	bint_two = bint_new32(2);
	bint_eight = bint_new32(8);
	bint_ten = bint_new32(10);

	return (1);
}

int bint_deinit() {
	
	bint_set64 = NULL;
	bint_set32 = NULL;
	bint_set16 = NULL;

	bint_add = NULL;
	bint_sub = NULL;
	bint_mult = NULL;
	bint_div = NULL;
	bint_mod = NULL;
	bint_divmod = NULL;

	bint_add_dst = NULL;
	bint_sub_dst = NULL;
	bint_mult_dst = NULL;
	bint_div_dst = NULL;
	bint_mod_dst = NULL;
	bint_divmod_dst = NULL;

	free(bint_zero);
	free(bint_two);
	free(bint_ten);

	return (1);
}