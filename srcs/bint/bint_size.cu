#include "bint.h"

#define _BINT_DEFAULT_SIZE (sizeof(int) * 1024)

unsigned int _default_size = _BINT_DEFAULT_SIZE;

void bint_set_default_size(unsigned int size) {
	_default_size = size;
}

unsigned int bint_get_default_size(void) {
	return (_default_size);
}

void bint_reset_default_size(void) {
	_default_size = _BINT_DEFAULT_SIZE;
}