#include "bint.h"

//the index of the first and last bit of a word
#define FIRST_WORD_BIT (0)
#define LAST_WORD_BIT (sizeof(int) * 8 - 1)

static void bint_shift_left_once(t_bint *dst) {
	unsigned int *ptr = dst->words + dst->size - 1;
	unsigned int *end = dst->last_word_set == dst->words ? dst->words : dst->last_word_set - 1;
	int reminder = 0;
	while (ptr > end) {
		//will it overflow: '* 1101' -> '* 1010' -> '*1 1010'
		int next_reminder = BITSET(*ptr, LAST_WORD_BIT);

		//operate the shift
		*ptr = *ptr << 1;

		//if there was a previous overflow, set the first bit
		if (reminder) {
			SETBIT(*ptr, FIRST_WORD_BIT);
		}
		reminder = next_reminder;
		--ptr;
	}
}

static void bint_shift_right_once(t_bint *dst) {
	unsigned int *ptr = dst->last_word_set;
	unsigned int *end = dst->words + dst->size;
	int reminder = 0;
	while (ptr < end) {
		//will it overflow: '1101 *' -> '0110 *' -> '0110 1*'
		int next_reminder = BITSET(*ptr, FIRST_WORD_BIT);

		//operate the shift
		*ptr = *ptr >> 1;

		//if there was a previous overflow, set the last bit
		if (reminder) {
			SETBIT(*ptr, LAST_WORD_BIT);
		}
		reminder = next_reminder;
		--ptr;
	}
}

static t_bint *bint_shift_dst_raw(t_bint **dst, t_bint *integer, unsigned int n, void (*shift_function)(t_bint *)) {

	//the pointer to store the result
	t_bint *r = bint_ensure_size(dst, integer->size);

	//if allocation failed...
	if (r == NULL) {
		return (NULL);
	}

	//copy the integer to shift
	bint_copy(r, integer);

	//shift it n times
	unsigned int i;
	for (i = 0 ; i < n ; i++) {
		shift_function(r);
	}

	return (r);
}

t_bint *bint_shift_left(t_bint *i, int n) {
	return (bint_shift_left_dst(NULL, i, n));
}

t_bint *bint_shift_left_dst(t_bint **dst, t_bint *i, int n) {

	//if i is 0, return 0
	if (i == NULL || i->sign == 0) {
		return (NULL);
	}

	//the shift function to use (left or right)
	void (*shift_function)(t_bint *);
	//the number of time to shift
	unsigned int times;

	//if n is negative, then we shift right -n times
	if (n < 0) {
		times = -n;
		shift_function = bint_shift_right_once;
	} else {
		//else n is positive, we shift left n times
		times = n;
		shift_function = bint_shift_left_once;
	}
	return (bint_shift_dst_raw(dst, i, times, shift_function));
}


t_bint *bint_shift_right(t_bint *i, int n) {
	return (bint_shift_left_dst(NULL, i, n));
}

t_bint *bint_shift_right_dst(t_bint **dst, t_bint *i, int n) {

	//if i is 0, return 0
	if (bint_is_zero(i)) {
		return (BINT_ZERO);
	}

	//the shift function to use (left or right)
	void (*shift_function)(t_bint *);
	//the number of time to shift
	unsigned int times;

	//if n is negative, then we shift left -n times
	if (n < 0) {
		times = -n;
		shift_function = bint_shift_left_once;
	} else {
		//else n is positive, we shift right n times
		times = n;
		shift_function = bint_shift_right_once;
	}
	return (bint_shift_dst_raw(dst, i, times, shift_function));
}