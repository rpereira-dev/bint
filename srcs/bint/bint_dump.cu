#include "bint.h"

void bint_dump(t_bint *i) {

	if (i == NULL || i->sign == 0) {
		printf("bint: {NULL or 0}");
	} else {
		char *sign = i->sign == -1 ? "-" : i->sign == 1 ? "+" : NULL;
		printf("bint: {size: %u, sign: %2d, words: %s", i->size, i->sign, sign);
		bdump(i->last_word_set, (i->size - (i->last_word_set - i->words)) * 4);
	}
}
