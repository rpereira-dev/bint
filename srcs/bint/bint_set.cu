#include "bint.h"

void bint_set64(t_bint *dst, long long int i) {

}

void bint_set32(t_bint *dst, int i) {

	//if we are setting a zero
	if (i == 0) {
		dst->sign = 0;
		return ;
	}

	//set the sign
	if (i < 0) {
		dst->sign = -1;
		i = -i;
	} else {
		dst->sign = 1;
	}

	//set other bits to 0
	memset(dst->words, 0, (dst->size - 1) * sizeof(int));
	
	//set the value
	int *addr = (int*)(dst->words + dst->size - 1);
	*addr = i;
	dst->last_word_set = dst->words + dst->size - 1;
}

void bint_set16(t_bint *dst, short i) {
	
}

void bint_set8(t_bint *dst, char i) {

}
