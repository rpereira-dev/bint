#include "bint.h"

t_bint *bint_set(t_bint **dst, int i) {

	//ensure dst capacity
	t_bint * r = bint_ensure_size(dst, bint_get_default_size());
	if (r == NULL) {
		return (NULL);
	}

	//if we are setting a zero
	if (i == 0) {
		r->sign = 0;
		r->wordset = 0;
		return (r);
	}

	//set the sign
	if (i < 0) {
		r->sign = -1;
		i = -i;
	} else {
		r->sign = 1;
	}

	//set other bits to 0
	memset(r->words, 0, (r->size - 1) * sizeof(int));
	
	//set the value
	int *addr = (int*)(r->words + r->size - 1);
	*addr = i;
	r->wordset = 1;
	return (r);
}
