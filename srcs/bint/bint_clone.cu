#include "bint.h"

/** no endian issues */
t_bint *bint_clone(t_bint *src) {

	if (src == NULL || src->sign == 0) {
		return (NULL);
	}

	t_bint *dst = bint_new(src->size);
	if (dst == NULL) {
		return (NULL);
	}
	dst->sign = src->sign;
	memcpy(dst->words, src->words, src->size * sizeof(int));
	dst->last_word_set = dst->words + (src->last_word_set - src->words);
	return (dst);
}