#include "bint.h"

/** no endian issues */
t_bint *bint_clone(t_bint *src) {

	if (bint_is_zero(src)) {
		return (BINT_ZERO);
	}

	t_bint *dst = bint_new(src->size);
	if (dst == NULL) {
		return (NULL);
	}
	
	dst->sign = src->sign;
	memcpy(dst->words, src->words, src->size * sizeof(int));
	dst->wordset = src->wordset;
	return (dst);
}