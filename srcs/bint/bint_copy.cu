#include "bint.h"

/** assumre that dst->size >= src->size */
void bint_copy(t_bint *dst, t_bint *src) {

	unsigned int src_word_set = src->words + src->size - src->last_word_set;
	dst->sign = src->sign;
	dst->last_word_set = dst->words + dst->size - src_word_set;
	memcpy(dst->last_word_set, src->last_word_set, src_word_set * sizeof(int));
}
