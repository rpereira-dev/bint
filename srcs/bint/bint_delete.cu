#include "bint.h"

void bint_delete(t_bint **dst) {
	if (dst == NULL || *dst == NULL) {
		return ;
	}
	free(*dst);
	*dst = NULL;
}