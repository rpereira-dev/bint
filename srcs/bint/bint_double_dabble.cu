#include "bint.h"

//the index of the first and last bit of a word
#define FIRST_WORD_BIT (0)
#define LAST_WORD_BIT (sizeof(int) * 8 - 1)

static void bint_shift_left_once(unsigned int *addr, size_t len) {
	unsigned int *ptr = addr + len;
	unsigned int *end = addr;
	int reminder = 0;
	while (ptr > end) {
		//will it overflow: '* 1101' -> '* 1010' -> '*1 1010'
		unsigned int next_reminder = BITSET(*ptr, LAST_WORD_BIT);

		//operate the shift
		*ptr = *ptr << 1;

		//if there was a previous overflow, set the first bit
		if (reminder) {
			SETBIT(*ptr, FIRST_WORD_BIT);
		}
		reminder = next_reminder;
		--ptr;
	}
}

void *bint_double_dabble(t_bint *i) {

	//if zero, return NULL
	if (i == BINT_ZERO || i->sign == 0) {
		return (BINT_ZERO);
	}

	//else calculate the bit sizes
	size_t nbits = i->size * sizeof(unsigned int);
	size_t nbits_bcd = nbits + 4 * nbits / 3;
	size_t nbytes_bcd = nbits_bcd / 8 + (nbits_bcd % 8 != 0);
	if (nbytes_bcd % sizeof(int) != 0) {
		 nbytes_bcd += (sizeof(int) - nbytes_bcd % sizeof(int));
	}

	//alocate bcd dst
	char *bcd = (char*)malloc(nbytes_bcd);
	if (bcd == NULL) {
		return (NULL);
	}

	//prepare the bcd pointer
	size_t wordset = i->words + i->size - i->last_word_set;
	size_t byteset = wordset * sizeof(int);
	memcpy(bcd + nbytes_bcd - byteset, i->last_word_set, byteset);
	memset(bcd, 0, nbytes_bcd - byteset);

	
}