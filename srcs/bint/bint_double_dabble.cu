#include "bint.h"

/** a function which shift left the 'len' byte at addr 'addr' */
static void bcd_shift_left_once(unsigned char *addr, size_t len) {
	unsigned char *ptr = addr + len - 1;
	unsigned char *end = addr;
	unsigned char reminder = 0;
	while (ptr >= end) {

		//check overflow (if last bit is set, then it will overflow)
		unsigned char next_reminder = *ptr & (1 << 7);

		//operate the shift
		*ptr = *ptr << 1;

		//if there was a previous overflow, set the first bit
		if (reminder) {
			*ptr = *ptr | 1;
		}
		reminder = next_reminder;
		--ptr;
	}
}

/** interpret the given address has an integer 32 bits array, and swap the endian of each integer */
void bint_bcd_swap_endian(void *addr, unsigned int nword) {
	unsigned int *words = (unsigned int*)addr;
	int i;

	for (i = 0 ; i < nword ; i++) {
		unsigned int n = words[i];
		words[i] = ((n >> 24) & 0xff) | ((n << 8) & 0xff0000) | ((n >> 8) & 0xff00) | ((n << 24) & 0xff000000);
	}
}

/** implementation based on this document: http://www.tkt.cs.tut.fi/kurssit/1426/S12/Ex/ex4/Binary2BCD.pdf */
char *bint_to_bcd(t_bint *i) {

	//double dabble implementation

	//if zero, return NULL
	if (i == BINT_ZERO || i->sign == 0) {
		return (strdup(""));
	}

	//calculate the integer total size
	size_t wordset = i->words + i->size - i->last_word_set;
	size_t byteset = wordset * sizeof(unsigned int);

	//else calculate the bit sizes
	size_t bitset = byteset * 8;

	//total number of bits for the bcd storage
	size_t nbits_bcd = bitset + 4 * bitset / 3;
	size_t nbytes_bcd = nbits_bcd / 8 + (nbits_bcd % 8 != 0);
	if (nbytes_bcd % sizeof(int) != 0) {
		 nbytes_bcd += (sizeof(int) - nbytes_bcd % sizeof(int));
	}

	//alocate bcd storage
	unsigned char *bcd = (unsigned char*)malloc(nbytes_bcd);
	if (bcd == NULL) {
		return (NULL);
	}

	//prepare the bcd pointer
	memcpy(bcd + nbytes_bcd - byteset, i->last_word_set, byteset);
	memset(bcd, 0, nbytes_bcd - byteset);

	//printf("bcd after copy: ", 0), bdump(bcd, nbytes_bcd), printf("\n");

	//swap the endian so we always work in little endian
	if (endianness() == BIG_ENDIAN) {
		bint_bcd_swap_endian(bcd + nbytes_bcd - byteset, wordset);
	}

	//printf("bcd endian fix: ", 0), bdump(bcd, nbytes_bcd), printf("\n");

	//shift so last set bit is right before first column
	unsigned char *bcd_begin = bcd + nbytes_bcd - byteset;
	//the address of the first bit set
	size_t bits_begin = 0;
	//the address of the last bit set
	size_t bits_end = 0;
	while (!(*bcd_begin & (1 << 7))) {

		//do the shift
		bcd_shift_left_once(bcd_begin, byteset);
		++bits_end;
	}

	//the address of the last bit set
	bits_end = bitset - bits_end;

	//total number of bits to shift
	size_t nbits = bits_end - bits_begin;

	// bits counter in column
	size_t bits = 0;

	//number of column set
	size_t bytes_in_column = 0;

	//printf("bcd initial   : ", bits), bdump(bcd, nbytes_bcd), printf("\n");

	//for each bits
	while (true) {

		//total number of byte hold by the bcd pointer (+ 1 so we handle overflow on shifting)
		size_t bytes_to_shift = byteset + bytes_in_column + 1;
		//where the shift should end
		unsigned char *endshift = bcd + nbytes_bcd - bytes_to_shift;
		//do the shift
		bcd_shift_left_once(endshift, bytes_to_shift);

		//increment number of bits in column
		++bits;
		if (*(bcd + nbytes_bcd - byteset - bytes_in_column - 1)) {
			++bytes_in_column;
		}

		//printf("bcd shift %4d: ", bits), bdump(bcd, nbytes_bcd), printf("\n");

		if (bits >= nbits) {
			break ;
		}


		//for each bcd byte column set
		int i;
		for (i = 0; i < bytes_in_column; i++) {
			//get the byte
			unsigned char *byteaddr = bcd + nbytes_bcd - byteset - 1 - i;

			//columns (4bits of the byte) can be : 0000 / 0001 / 0010 / 0011 / 1000 / 0101 / 0110 / 0111....
			char col;

			//get the first column
			col = *byteaddr & 0xF;

			//if first column >= 5
			if (col >= 5) {
				unsigned char *add_addr = byteaddr;
				unsigned char to_add = 3;

				while (to_add != 0) {
					unsigned char can_hold = 255 - *add_addr;
					if (can_hold > 3) {
						can_hold = 3;
					}
					to_add -= can_hold;
					*add_addr += can_hold;
					--add_addr;
				}
				//printf("bcd +3(1) %4d: ", bits), bdump(bcd, nbytes_bcd), printf("\n");
			}

			//get the second column
			col = (*byteaddr & 0xF0) >> 4;

			//if the second column >= 5
			if (col >= 5) {
				if (col <= 12) {
					//printf("bcd +3(2) %4d: ", bits), bdump(bcd, nbytes_bcd), printf("\n");
					*byteaddr = (*byteaddr & 0xF) | ((col + 3) << 4);
					//printf("bcd +3(2) %4d: ", bits), bdump(bcd, nbytes_bcd), printf("\n");
				} else {
					puts("warning: bcd algorythm error (2nd 4 bytes > 12)");
				}
			}
		}
	}


	//reallocate a memory space to hold the final bcd number
	char *bcd_final = (char*)malloc(bytes_in_column + 1);
	if (bcd_final == NULL) {
		free(bcd);
		return (NULL);
	}

	//copy the data
	memcpy(bcd_final, bcd_begin - bytes_in_column, bytes_in_column);
	//last char
	bcd_final[bytes_in_column] = 0xFF;
	//printf("%d\n", bytes_in_column);

	free(bcd);
	return (bcd_final);
}

static char bcd_char_to_decimal(char c) {

	c = c + '0';
	return ((c >= '0' && c <= '9') ? c : '?');
}

/** convert the bcd number to a decimal string */
char *bcd_to_str(char *bcd) {
	if (bcd == NULL) {
		return (NULL);
	}
	//find end of the bcd string
	char *end = strchr(bcd, 0xFF);
	if (end == NULL) {
		return (NULL);
	}
	//allocate the new string
	size_t bcdlength = end - bcd;
	size_t strlength = bcdlength * 2;
	char *str = (char*)malloc(sizeof(char) * (strlength + 1));
	if (str == NULL) {
		return (NULL);
	}
	//null terminate the string
	str[strlength] = 0;

	//handle first two char (so there is no '0' heading the number)
	size_t i = 0;
	size_t j = 0;

	while (i < bcdlength) {
		
		//get bcd chars
		char c1 = ((bcd[i] & 0xF0) >> 4);
		char c2 = (bcd[i] & 0xF);

		//add the char
		str[j++] = bcd_char_to_decimal(c1);
		str[j++] = bcd_char_to_decimal(c2);
		++i;
	}

	return (str);
}