#include "bint.h"

static unsigned int clamp_size(unsigned int size) {
	if (size == 0) {
		size = sizeof(int);
	} else if (size % sizeof(int) != 0) {
		size += (sizeof(int) - size % sizeof(int));
	}
	return (size);
}

t_bint *bint_new(unsigned int size) {

	//so the size is > 0 and multiple of sizeof(int)
	size = clamp_size(size);

	//allocate memory space
	t_bint *i = (t_bint*)malloc(sizeof(t_bint) + size);
	if (i == NULL) {
		return (NULL);
	}

	//assign size
	i->size = size;
	i->sign = 0;
	i->bits = (char*)(i + 1);

	//return it
	return (i);
}

/** 8bits version new and set */
t_bint *bint_new8(char value) {
	t_bint *i = bint_new(sizeof(char));
	if (i == NULL) {
		return (NULL);
	}
	bint_set8(i, value);
	return (i);
}

/** 16bits version new and set */
t_bint *bint_new16(short value) {
	t_bint *i = bint_new(sizeof(short));
	if (i == NULL) {
		return (NULL);
	}
	bint_set16(i, value);
	return (i);
}

/** 32bits version new and set */
t_bint *bint_new32(int value) {
	t_bint *i = bint_new(sizeof(int));
	if (i == NULL) {
		return (NULL);
	}
	bint_set32(i, value);
	return (i);
}

/** 64bits version new and set */
t_bint *bint_new64(long int value) {
	t_bint *i = bint_new(sizeof(long int));
	if (i == NULL) {
		return (NULL);
	}
	bint_set64(i, value);
	return (i);
}