#include "bint.h"

t_bint *bint_new(unsigned int size) {

	//so the size is > 0 and multiple of sizeof(int)
	if (size == 0) {
		size = 1;
	}

	//allocate memory space
	t_bint *i = (t_bint*)malloc(sizeof(t_bint) + size * sizeof(int));
	if (i == NULL) {
		return (NULL);
	}

	//assign size
	i->size = size;
	i->sign = 0;
	i->words = (unsigned int*)(i + 1);
	i->last_word_set = i->words + size;

	//return it
	return (i);
}

/** 8bits version new and set */
t_bint *bint_new8(char value) {
	t_bint *i = bint_new(sizeof(char));
	if (i == NULL) {
		return (NULL);
	}
	bint_set8(i, value);
	return (i);
}

/** 16bits version new and set */
t_bint *bint_new16(short value) {
	t_bint *i = bint_new(sizeof(short));
	if (i == NULL) {
		return (NULL);
	}
	bint_set16(i, value);
	return (i);
}

/** 32bits version new and set */
t_bint *bint_new32(int value) {
	t_bint *i = bint_new(sizeof(int));
	if (i == NULL) {
		return (NULL);
	}
	bint_set32(i, value);
	return (i);
}

/** 64bits version new and set */
t_bint *bint_new64(long int value) {
	t_bint *i = bint_new(sizeof(long long int));
	if (i == NULL) {
		return (NULL);
	}
	bint_set64(i, value);
	return (i);
}