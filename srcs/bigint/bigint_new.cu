#include "bigint.h"

t_bigint *bigint_new(unsigned int bytes) {

	if (bytes < 4) {
		bytes = 4;
	}

	t_bigint *i = (t_bigint*)malloc(sizeof(t_bigint) + bytes);
	if (i == NULL) {
		return (NULL);
	}
	i->bytes = bytes;
	return (i);
}