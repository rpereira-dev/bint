#include "bigint.h"

void bigint_set32(t_bigint *i, int value) {

	char *bytes = (char*)(i + 1);

	if (value < 0) {
		memset(bytes, -1, i->bytes - 4);
	}

	value = ensure_int_endianess(value);

	memcpy(bytes + i->bytes - 4, &value, 4);
}